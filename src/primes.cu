
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

typedef unsigned long __largeuint_t;

// fun stuff to mess with
// if you can add commandline params
// to replace recompiling every time,
// ill merge it in

char calcSingle = 1;  // whether to calculate the single input (1) or the range (0)

__largeuint_t singleInput = 50000000021;  // single input number to test

__largeuint_t start = 20;  // start of calculation range
__largeuint_t end   = 20;  // end of calculation range

// end fun stuff, start boring stuff

// Kernel function
__global__
void kernel(__largeuint_t input, __largeuint_t globalMaxCompare, int *globalIsPrime) {
    __largeuint_t index  = blockIdx.x * blockDim.x + threadIdx.x * 2 + 3;
    __largeuint_t stride = blockDim.x * gridDim.x * 2;

    if (index == 0 || index == 1) {
        printf("INDEX IS %lu! SOMETHING HAS GOME WRONG!\n\n", index, blockIdx.x, blockDim.x, threadIdx.x);
        *globalIsPrime = 0;
        return;
    }

    for (__largeuint_t compare = index; compare < globalMaxCompare && globalIsPrime; compare += stride) {
        if (input % compare == 0) {
            printf("%lu is devisable by %lu\n", input, compare);
            *globalIsPrime = 0;
            break;
        }
    }
}

int isPrime(__largeuint_t globalInput) {
    __largeuint_t globalMaxCompare = globalInput / 3;
    int *globalIsPrime;

    if (globalInput % 2 == 0) {
        printf("%lu is devisable by 2\n", globalInput);
        return 0;
    }

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&globalIsPrime, sizeof(int));

    *globalIsPrime = 1;

    // Run kernel on 1M elements on the GPU
    int blockSize = 512;
    int numBlocks = (globalMaxCompare/2 + blockSize - 1) / blockSize;
    kernel<<<numBlocks, blockSize>>>(globalInput, globalMaxCompare, globalIsPrime);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    int output = *globalIsPrime;

    // Free memory
    hipFree(globalIsPrime);

    return output;
}

int main() {
    if (calcSingle) {
        printf("Begining calculation of %lu\n", singleInput);
        if (isPrime(singleInput) == 1) {
            printf("%lu is prime\ndone\n", singleInput);
        } else {
            printf("%lu is not prime\ndone\n", singleInput);
        }
    } else {
        printf("Calculating primes from %lu to %lu\n", start, end);
        for (__largeuint_t i = start; i < end; i++)
        {
            if (isPrime(i) == 1) {
                printf("%lu\n", i);
            }
        }
    }

    return 0;
}
